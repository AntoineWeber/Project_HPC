#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "kernels.cuh"
#include "brute_force.h"


__global__ void initialize_particles(float *x_pos, float *y_pos, float *x_vel, float *y_vel, float *x_acc, float *y_acc, float *mass)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int stride = gridDim.x * blockDim.x;
    int offset = 0;

    // to initialize the cuda rand
    hiprandState state;
    hiprand_init(clock64(), i, 0, &state);

    while (i + offset < N_PARTICLES)
    {
        mass[i] = (float)PARTICLE_MASS;

        x_pos[i] = hiprand_uniform(&state)*GRID_MAX*2 + GRID_MIN;
        y_pos[i] = hiprand_uniform(&state)*GRID_MAX*2 + GRID_MIN;

        // set velocity to 0
        x_vel[i] = 0.0;
        y_vel[i] = 0.0;

        // set acceleration to 0
        x_acc[i] = 0.0;
        y_acc[i] = 0.0;

        offset += stride;
    }
}


__global__ void compute_forces(float *x_pos, float *y_pos, float *x_vel, float *y_vel, float *x_acc, float *y_acc, float *mass)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int stride = gridDim.x * blockDim.x;
    int offset = 0;

    float fx=0;
    float fy=0;
    // loop on given particles if not enough threads
    while (i + offset < N_PARTICLES)
    {
        //printf(" %d ", offset);
        for (int j=0; j<N_PARTICLES; j++)
        {
            if (j != i)
            {
                // compute forces in x and y
                fx += (G*mass[i+offset]*mass[j]*(x_pos[j]-x_pos[i+offset]))/(sqrt((x_pos[j]-x_pos[i+offset])*(x_pos[j]-x_pos[i+offset])));
                fy += (G*mass[i+offset]*mass[j]*(y_pos[j]-y_pos[i+offset]))/(sqrt((y_pos[j]-y_pos[i+offset])*(y_pos[j]-y_pos[i+offset])));
            }
        }

        // F = ma -> a = F/m
        x_acc[i+offset] = fx / mass[i+offset];
        y_acc[i+offset] = fy / mass[i+offset];
        
        offset += stride;
    }
    __syncthreads();
    
    // have to define this second loop otherwise would move particles before having computed the force for all of them
    offset = 0;
    while (i + offset < N_PARTICLES)
    {
        x_pos[i+offset] += 0.5*x_acc[i+offset]*TIMESTEP*TIMESTEP + x_vel[i+offset]*TIMESTEP;
        y_pos[i+offset] += 0.5*y_acc[i+offset]*TIMESTEP*TIMESTEP + y_vel[i+offset]*TIMESTEP;

        x_vel[i+offset] += x_acc[i+offset]*TIMESTEP;
        y_vel[i+offset] += y_acc[i+offset]*TIMESTEP;

        offset+=stride;
    }
    __syncthreads();
}
